
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU! %d\n", threadIdx.x);
}

int main() {
    printf("Hello world from CPU!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}