#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using shared memory to transpose rectangular thread coordinates
 * of a CUDA grid into a global memory array. Different kernels below
 * demonstrate performing reads and writes with different ordering, as well as
 * optimizing using memory padding.
 */

#define BDIMX 32
#define BDIMY 16
#define IPAD  2

void printData(const char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%4d", in[i]);
        fflush(stdout);
    }

    printf("\n\n");
}

__global__ void setRowReadRow(int *out)
{
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int *out)
{
    __shared__ int tile[BDIMX][BDIMY];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.x][threadIdx.y] = idx;
    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setColReadCol2(int *out)
{
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int rowIdx = idx % blockDim.y;
    unsigned int colIdx = idx / blockDim.y;

    tile[rowIdx][colIdx] = idx;
    __syncthreads();

    out[idx] = tile[rowIdx][colIdx];
}

__global__ void setRowReadCol(int *out)
{
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int rowIdx = idx % blockDim.y;
    unsigned int colIdx = idx / blockDim.y;

    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();

    out[idx] = tile[rowIdx][colIdx];
}

__global__ void setRowReadColPad(int *out)
{

}

__global__ void setRowReadColDyn(int *out)
{
    extern __shared__ int tile[];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int rowIdx = idx % blockDim.y;
    unsigned int colIdx = idx / blockDim.y;

    unsigned int tileIdx = rowIdx * blockDim.x + colIdx;

    tile[idx] = idx;
    __syncthreads();

    out[idx] = tile[tileIdx];
}

__global__ void setRowReadColDynPad(int *out)
{

   
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CHECK(hipDeviceGetSharedMemConfig ( &pConfig ));
    printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

    // set up array size
    int nx = BDIMX;
    int ny = BDIMY;

    bool iprintf = 0;

    if (argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    dim3 grid  (1, 1);
    printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
            block.y);

    // allocate device memory
    int *d_C;
    CHECK(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setRowReadRow       ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setColReadCol       ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol2<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setColReadCol2      ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setRowReadCol       ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMX*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setRowReadColDyn    ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setRowReadColPad    ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDynPad<<<grid, block, (BDIMX + IPAD)*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("setRowReadColDynPad ", gpuRef, nx * ny);

    // free host and device memory
    CHECK(hipFree(d_C));
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}