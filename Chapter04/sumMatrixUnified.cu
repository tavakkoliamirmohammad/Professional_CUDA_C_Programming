#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates using explicit CUDA memory transfer to implement
 * matrix addition. This code contrasts with sumMatrixGPUManaged.cu, where CUDA
 * managed memory is used to remove all explicit memory transfers and abstract
 * away the concept of physicall separate address spaces.
 */

void initialData(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }

    return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (!match)
    {
        printf("Arrays do not match.\n\n");
    }
}

// grid 2D block 2D
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx,
                             int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting ", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx, ny;
    int ishift = 12;

    if  (argc > 1) ishift = atoi(argv[1]);

    nx = ny = 1 << ishift;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipMallocManaged((void **)&h_A, nBytes));
    CHECK(hipMallocManaged((void **)&h_B, nBytes));
    CHECK(hipMallocManaged((void **)&hostRef, nBytes));
    CHECK(hipMallocManaged((void **)&gpuRef, nBytes));

    // initialize data at host side
    double iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    double iElaps = seconds() - iStart;

    printf("initialization: \t %f sec\n", iElaps);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;
    printf("sumMatrix on host:\t %f sec\n", iElaps);

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // warm-up kernel, with unified memory all pages will migrate from host to
    // device
    sumMatrixGPU<<<grid, block>>>(h_A, h_B, gpuRef, 1, 1);

    // after warm-up, time with unified memory
    iStart = seconds();

    sumMatrixGPU<<<grid, block>>>(h_A, h_B, gpuRef, nx, ny);

    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrix on gpu :\t %f sec <<<(%d,%d), (%d,%d)>>> \n", iElaps,
            grid.x, grid.y, block.x, block.y);

    // check kernel error
    CHECK(hipGetLastError());

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(h_A));
    CHECK(hipFree(h_B));
    CHECK(hipFree(hostRef));
    CHECK(hipFree(gpuRef));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}